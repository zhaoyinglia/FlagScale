#include "hip/hip_runtime.h"
// 2025 - Modified by MetaX Integrated Circuits (Shanghai) Co., Ltd. All Rights Reserved.
/*
 * Adapted from
 * https://github.com/NVIDIA/FasterTransformer/blob/release/v5.3_tag/src/fastertransformer/kernels/decoder_masked_multihead_attention/decoder_masked_multihead_attention_template.hpp
 * Copyright (c) 2023, The vLLM team.
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "attention_kernels.cuh"

#ifndef USE_ROCM
  #define WARP_SIZE 32
#else
  #define WARP_SIZE warpSize
#endif

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

#define LAUNCH_PAGED_ATTENTION_V1(HEAD_SIZE)                                \
  VLLM_DevFuncAttribute_SET_MaxDynamicSharedMemorySize(                     \
      ((void*)vllm::paged_attention_v1_kernel<T, CACHE_T, HEAD_SIZE,        \
                                              BLOCK_SIZE, NUM_THREADS,      \
                                              KV_DTYPE, IS_BLOCK_SPARSE>),  \
      shared_mem_size);                                                     \
  vllm::paged_attention_v1_kernel<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE,        \
                                  NUM_THREADS, KV_DTYPE, IS_BLOCK_SPARSE>   \
      <<<grid, block, shared_mem_size, stream>>>(                           \
          out_ptr, query_ptr, key_cache_ptr, value_cache_ptr, num_kv_heads, \
          scale, block_tables_ptr, seq_lens_ptr, max_num_blocks_per_seq,    \
          alibi_slopes_ptr, q_stride, kv_block_stride, kv_head_stride,      \
          k_scale_ptr, v_scale_ptr, tp_rank, blocksparse_local_blocks,              \
          blocksparse_vert_stride, blocksparse_block_size,                  \
          blocksparse_head_sliding_step);

#define LAUNCH_PAGED_ATTENTION_V1_32N(HEAD_SIZE)                                \
  VLLM_DevFuncAttribute_SET_MaxDynamicSharedMemorySize(                     \
      ((void*)vllm::paged_attention_v1_32N_kernel<T, CACHE_T, HEAD_SIZE,        \
                                              BLOCK_SIZE, NUM_THREADS,      \
                                              KV_DTYPE, IS_BLOCK_SPARSE>),  \
      shared_mem_size);                                                     \
  vllm::paged_attention_v1_32N_kernel<T, CACHE_T, HEAD_SIZE, BLOCK_SIZE,        \
                                  NUM_THREADS, KV_DTYPE, IS_BLOCK_SPARSE>   \
      <<<grid, block, shared_mem_size, stream>>>(                           \
          out_ptr, query_ptr, key_cache_ptr, value_cache_ptr, num_kv_heads, \
          scale, block_tables_ptr, seq_lens_ptr, max_num_blocks_per_seq,    \
          alibi_slopes_ptr, q_stride, kv_block_stride, kv_head_stride,      \
          k_scale_ptr, v_scale_ptr, tp_rank, blocksparse_local_blocks,                      \
          blocksparse_vert_stride, blocksparse_block_size,                  \
          blocksparse_head_sliding_step, 1, num_heads);

template< typename scalar_t>
__global__ void reshape_k_layout_new(scalar_t * __restrict__ k_buffer, scalar_t* k_output,int num_blocks,int num_kv_heads, int head_size,int block_size, int x,int dst_x) {
  int k_head_stride = head_size * block_size;
  scalar_t *ptr_k_buffer = k_buffer + blockIdx.x * k_head_stride;
  scalar_t *ptr_output = k_output + blockIdx.x * k_head_stride;
  for(int t = threadIdx.x; t < k_head_stride; t += blockDim.x) {
    int heightId = t / (block_size * dst_x);
    int remain = t % (block_size * dst_x);
    int blockId = remain / dst_x;
    int wId = remain % dst_x;
    int inId = heightId * dst_x + wId;
    int in_y = inId / x;
    int in_x = inId % x;
    int inIndex = in_y  * block_size * x + blockId * x + in_x;
    ptr_output[t] = ptr_k_buffer[inIndex];
  }
}
// [num_blocks, num_kv_heads, head_size, block_size] -->   [num_blocks,  num_kv_heads, block_size,head_size]
template<typename scalar_t>
__global__ void reshape_v_layout(scalar_t * __restrict__ v_buffer, scalar_t* v_output,int num_blocks,int num_kv_heads, int head_size,int block_size) {
      int v_block_stride = head_size * block_size * num_kv_heads;
      int v_head_stride = head_size * block_size;
      scalar_t *ptr_in = v_buffer + blockIdx.x * v_block_stride;
      scalar_t *ptr_output = v_output + blockIdx.x * v_block_stride;
      for(int t = threadIdx.x; t < v_block_stride; t += blockDim.x) {
        int num_kv_headIdx = t / v_head_stride;
        int remain = t % v_head_stride;
        int headId_H = remain / block_size;
        remain = remain % block_size;
        int out_idx = num_kv_headIdx * head_size * block_size + remain * head_size + headId_H;
        ptr_output[out_idx] = ptr_in[t];
      }
}

template<
  typename CACHE_T,
  int BLOCK_SIZE>
void reshape_kv_cache(
  torch::Tensor& key_cache,
  torch::Tensor& value_cache,
  torch::Tensor& key_cache_new_layer,
  torch::Tensor& value_cache_new_layer,
  int num_seqs,
  int num_heads,
  int head_size,
  int num_kv_heads) {
  int kv_block_stride = key_cache.stride(0); // NU ,BLC ,HEAD, HEAD_DIM
  int kv_head_stride = key_cache.stride(1);

  CACHE_T* key_cache_ptr = reinterpret_cast<CACHE_T*>(key_cache.data_ptr());
  CACHE_T* value_cache_ptr = reinterpret_cast<CACHE_T*>(value_cache.data_ptr());
  CACHE_T* key_cache_tmp = reinterpret_cast<CACHE_T*>(key_cache_new_layer.data_ptr());
  CACHE_T* value_cache_tmp = reinterpret_cast<CACHE_T*>(value_cache_new_layer.data_ptr());

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  reshape_k_layout_new<CACHE_T><<<dim3(key_cache.size(0)*num_kv_heads,1,1),dim3(256,1,1),0,stream>>>(key_cache_ptr,key_cache_tmp,key_cache.size(0),num_kv_heads,head_size,BLOCK_SIZE,8,16);
  reshape_v_layout<CACHE_T><<<dim3(key_cache.size(0),1,1),dim3(256,1,1),0,stream>>>(value_cache_ptr,value_cache_tmp,key_cache.size(0),num_kv_heads,head_size,BLOCK_SIZE);
}
#define CALL_RESHAPE_LAUNCHER(CACHE_T, BLOCK_SIZE)       \
  reshape_kv_cache<CACHE_T, BLOCK_SIZE>( \
    key_cache,                                                               \
    value_cache,                                                             \
    key_cache_new_layer,                                                     \
    value_cache_new_layer,                                                   \
    num_seqs,\
    num_heads,\
    head_size,\
    num_kv_heads);

#define CALL_RESHAPE_BLOCK_SIZE(CACHE_T) \
  switch (block_size) {                                               \
    case 8:                                                           \
      CALL_RESHAPE_LAUNCHER(CACHE_T, 8);          \
      break;                                                          \
    case 16:                                                          \
      CALL_RESHAPE_LAUNCHER(CACHE_T, 16);         \
      break;                                                          \
    case 32:                                                          \
      CALL_RESHAPE_LAUNCHER(CACHE_T, 32);         \
      break;                                                          \
    default:                                                          \
      TORCH_CHECK(false, "Unsupported block size: ", block_size);     \
      break;                                                          \
  }
void page_reshape_kv_cache(
  torch::Tensor& key_cache,       // [num_blocks, num_heads, head_size/x, block_size, x]
  torch::Tensor& value_cache,     // [num_blocks, num_heads, head_size, block_size]
  torch::Tensor& key_cache_new_layer, //[num_blocks, num_heads, head_size/16, block_size, 16]
  torch::Tensor& value_cache_new_layer,//[num_blocks, num_heads, block_size, head_size]
  int64_t num_seqs,
  int64_t num_heads,
  int64_t head_size,
  int64_t num_kv_heads,               // [num_heads]
  int64_t block_size,
  const std::string& kv_cache_dtype) {
  if (kv_cache_dtype == "auto") {
    if (sizeof(key_cache.dtype())==4) {
      CALL_RESHAPE_BLOCK_SIZE(float);
    } else if (sizeof(key_cache.dtype()) == 2) {
      CALL_RESHAPE_BLOCK_SIZE(uint16_t);
    } else {
      TORCH_CHECK(false, "Unsupported data type: ", key_cache.dtype());
    }
  }  else {
    TORCH_CHECK(false, "Unsupported data type of kv cache: ", kv_cache_dtype);
  }
}


// TODO(woosuk): Tune NUM_THREADS.
template <typename T, typename CACHE_T, int BLOCK_SIZE,
          vllm::Fp8KVCacheDataType KV_DTYPE, bool IS_BLOCK_SPARSE,
          int NUM_THREADS = 256>
void paged_attention_v1_launcher(
    torch::Tensor& out, torch::Tensor& query, torch::Tensor& key_cache,
    torch::Tensor& value_cache, int num_kv_heads, float scale,
    torch::Tensor& block_tables, torch::Tensor& seq_lens, int max_seq_len,
    const c10::optional<torch::Tensor>& alibi_slopes, torch::Tensor& k_scale,
    torch::Tensor& v_scale, const int tp_rank, const int blocksparse_local_blocks,
    const int blocksparse_vert_stride, const int blocksparse_block_size,
    const int blocksparse_head_sliding_step) {
  int num_seqs = query.size(0);
  int num_heads = query.size(1);
  int head_size = query.size(2);
  int max_num_blocks_per_seq = block_tables.size(1);
  int q_stride = query.stride(0);  //num head head_dim
  int kv_block_stride = key_cache.stride(0);   // NU ,BLC ,HEAD, HEAD_DIM
  int kv_head_stride = key_cache.stride(1);

  int thread_group_size = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  assert(head_size % thread_group_size == 0);
  assert((head_size & 7) == 0);

  // NOTE: alibi_slopes is optional.
  const float* alibi_slopes_ptr = alibi_slopes ?
    reinterpret_cast<const float*>(alibi_slopes.value().data_ptr())
    : nullptr;

  T* out_ptr = reinterpret_cast<T*>(out.data_ptr());
  T* query_ptr = reinterpret_cast<T*>(query.data_ptr());
  CACHE_T* key_cache_ptr = reinterpret_cast<CACHE_T*>(key_cache.data_ptr());
  CACHE_T* value_cache_ptr = reinterpret_cast<CACHE_T*>(value_cache.data_ptr());
  int* block_tables_ptr = block_tables.data_ptr<int>();
  int* seq_lens_ptr = seq_lens.data_ptr<int>();
  const float* k_scale_ptr = reinterpret_cast<const float*>(k_scale.data_ptr());
  const float* v_scale_ptr = reinterpret_cast<const float*>(v_scale.data_ptr());

  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  int padded_max_seq_len = DIVIDE_ROUND_UP(max_seq_len, BLOCK_SIZE) * BLOCK_SIZE;
  int logits_size = padded_max_seq_len * sizeof(float);
  int V_VEC_SIZE = 16 / sizeof(CACHE_T);
  int NUM_V_VECS_PER_THREAD = head_size / V_VEC_SIZE;
  int NUM_COLS_PER_ITER = MAX(WARP_SIZE / NUM_V_VECS_PER_THREAD, 1);
  int outputs_size = NUM_WARPS * head_size * sizeof(float) * NUM_COLS_PER_ITER;
  // Python-side check in vllm.worker.worker._check_if_can_support_max_seq_len
  // Keep that in sync with the logic here!
  int shared_mem_size = std::max(logits_size, outputs_size);

  dim3 grid(num_heads, num_seqs, 1);
  dim3 block(NUM_THREADS);
  const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  switch (head_size) {
    // NOTE(woosuk): To reduce the compilation time, we only compile for the
    // head sizes that we use in the model. However, we can easily extend this
    // to support any head size which is a multiple of 16.
    case 32:
      LAUNCH_PAGED_ATTENTION_V1(32);
      break;
    case 64:
      LAUNCH_PAGED_ATTENTION_V1_32N(64);
      break;
    case 80:
      LAUNCH_PAGED_ATTENTION_V1(80);
      break;
    case 96:
      LAUNCH_PAGED_ATTENTION_V1(96);
      break;
    case 112:
      LAUNCH_PAGED_ATTENTION_V1(112);
      break;
    case 120:
      LAUNCH_PAGED_ATTENTION_V1(120);
      break;
    case 128:
      LAUNCH_PAGED_ATTENTION_V1_32N(128);
      break;
    case 160:
      LAUNCH_PAGED_ATTENTION_V1(160);
    case 192:
      LAUNCH_PAGED_ATTENTION_V1(192);
      break;
    case 256:
      LAUNCH_PAGED_ATTENTION_V1_32N(256);
      break;
    default:
      TORCH_CHECK(false, "Unsupported head size: ", head_size);
      break;
  }
}

#define CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE, KV_DTYPE, IS_BLOCK_SPARSE)  \
  paged_attention_v1_launcher<T, CACHE_T, BLOCK_SIZE, KV_DTYPE,              \
                              IS_BLOCK_SPARSE>(                              \
      out, query, key_cache, value_cache, num_kv_heads, scale, block_tables, \
      seq_lens, max_seq_len, alibi_slopes, k_scale, v_scale, tp_rank,        \
      blocksparse_local_blocks, blocksparse_vert_stride,                     \
      blocksparse_block_size, blocksparse_head_sliding_step);

#define CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE) \
  if (is_block_sparse) {                                                   \
    CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE, true);       \
  } else {                                                                 \
    CALL_V1_LAUNCHER(T, CACHE_T, BLOCK_SIZE, IS_FP8_KV_CACHE, false);      \
  }

// NOTE(woosuk): To reduce the compilation time, we omitted block sizes
// 1, 2, 4, 64, 128, 256.
#define CALL_V1_LAUNCHER_BLOCK_SIZE(T, CACHE_T, KV_DTYPE)         \
  switch (block_size) {                                           \
    case 8:                                                       \
      CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, 8, KV_DTYPE);         \
      break;                                                      \
    case 16:                                                      \
      CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, 16, KV_DTYPE);        \
      break;                                                      \
    case 32:                                                      \
      CALL_V1_LAUNCHER_SPARSITY(T, CACHE_T, 32, KV_DTYPE);        \
      break;                                                      \
    default:                                                      \
      TORCH_CHECK(false, "Unsupported block size: ", block_size); \
      break;                                                      \
  }

void paged_attention_v1(
    torch::Tensor& out,    // [num_seqs, num_heads, head_size]
    torch::Tensor& query,  // [num_seqs, num_heads, head_size]
    torch::Tensor&
        key_cache,  // [num_blocks, num_heads, head_size/x, block_size, x]
    torch::Tensor&
        value_cache,       // [num_blocks, num_heads, head_size, block_size]
    int64_t num_kv_heads,  // [num_heads]
    double scale,
    torch::Tensor& block_tables,  // [num_seqs, max_num_blocks_per_seq]
    torch::Tensor& seq_lens,      // [num_seqs]
    int64_t block_size, int64_t max_seq_len,
    const c10::optional<torch::Tensor>& alibi_slopes,
    const std::string& kv_cache_dtype, torch::Tensor& k_scale, torch::Tensor& v_scale,
    const int64_t tp_rank, const int64_t blocksparse_local_blocks,
    const int64_t blocksparse_vert_stride, const int64_t blocksparse_block_size,
    const int64_t blocksparse_head_sliding_step) {
  const bool is_block_sparse = (blocksparse_vert_stride > 1);

  DISPATCH_BY_KV_CACHE_DTYPE(query.dtype(), kv_cache_dtype,
                             CALL_V1_LAUNCHER_BLOCK_SIZE)
}

#undef WARP_SIZE
#undef MAX
#undef MIN
#undef DIVIDE_ROUND_UP
